#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "device_types.h"
#include <assert.h>


typedef unsigned char u_char;

static void HandleError(hipError_t err,
	const char* file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void matrixMelt(unsigned char* A, unsigned char* B, unsigned char* C, int start,
	int wA, int wB, int size, const float* alphaTable)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx > size)
	{
		return;
	}
	int aIdx = (idx /3) % 10;
	if (idx == 88)
	{
		for (size_t i = 0; i < 10; i++)
		{
			printf("%f ", alphaTable[i]);
		}
		printf("\n");
	}
	float alpha = aIdx*1.0 / wA;
	printf("alpha[%d] is %f, index in mat is %d\n", aIdx, alpha, idx);
	C[idx] = A[idx] * alpha + B[idx] * (1 - alpha);

}

void meltCall(int width, int height, int channels)
{
	u_char* dev_A = NULL;
	u_char* dev_B = NULL;
	u_char* dev_C = NULL;


	u_char A[10][30];
	u_char B[10][30];
	u_char C[10][30] = { 0 };
	for (int i = 0; i < 30; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			A[j][i] = 100;
			B[j][i] = 0;
		}
	}
	float* alphaTableVer = (float*)malloc(width * sizeof(float));
	float* alphaTableHor = (float*)malloc(height * sizeof(float));
	for (size_t i = 0; i < height; i++)
	{
		alphaTableVer[i] = 1 - i / (height * 1.0);
		printf("%f  ", alphaTableVer[i]);
	}
	printf("\n");
	for (size_t i = 0; i < width; i++)
	{
		alphaTableHor[i] = 1 - i / (width * 1.0);
		printf("%f  ", alphaTableHor[i]);
	}
	printf("\n");
	float* dev_alphaTableVer = NULL;
	float* dev_alphaTableHor = NULL;
	const int mallocSize = sizeof(u_char) * width * height * channels;
	HANDLE_ERROR(hipMalloc((void**)&dev_A, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_B, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_C, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_alphaTableVer, height * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_alphaTableHor, height * sizeof(float)));
	int sizeOfTableHor = sizeof(float) * height;
	HANDLE_ERROR(hipMemcpy(dev_alphaTableHor, alphaTableHor, sizeOfTableHor, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_A, A, mallocSize, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_B, B, mallocSize, hipMemcpyHostToDevice));

	dim3 threadsPerBlock(32, 32);
	int gridCols = (30 + 32 - 1) / threadsPerBlock.x;
	int gridRows = (10 + 32 - 1) / threadsPerBlock.y;
	dim3 blocksPerGrid(gridCols, gridRows);

	matrixMelt <<<blocksPerGrid, threadsPerBlock >>> (dev_A, dev_B, dev_C, 0, 10, 10, mallocSize,
		dev_alphaTableHor);
	HANDLE_ERROR(hipGetLastError());
	hipMemcpy(C, dev_C, mallocSize, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < 10; i++)
	{
		for (size_t j = 0; j < 30; j++)
		{
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}

	free(alphaTableHor);
	free(alphaTableVer);

}

int main()
{
	meltCall(10, 10, 3);
}
