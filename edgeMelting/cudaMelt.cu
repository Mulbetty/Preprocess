#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "device_types.h"
#include <assert.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "cudaMelt.cuh"



__global__ void matrixMeltTable(unsigned char* A, unsigned char* B, unsigned char* C, int start,
int wA, int wB, int size,  float* alphaTable)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by*gridDim.x+blockIdx.x)*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x;
	if (idx > size)
	{
		return;
	}
	float alpha = alphaTable[idx/3];
	int idxIndex = idx / 3;

	int alphaIdx = idxIndex % 10;
	
	//printf("alpha[%d] is %f\n", idx, alpha);
	//C[idx] = A[idx] * alphaTable[alphaIdx] + B[idx] * (1 - alpha);
	C[idx] = A[idx]*alphaTable[alphaIdx]+ (int)B[idx];
	printf("��ǰ�߳�����idxΪ��%d, ��õ�alphaIdxΪ��%d, A[%d]��ֵΪ��%d, C[%d]��ֵΪ%d, alphaTable[alphaIdx] = %f\n", idxIndex, alphaIdx,idx, A[idx], idx, C[idx], alphaTable[alphaIdx]);
	
}

__global__ void matrixMeltHor(unsigned char* A, unsigned char* B, unsigned char* C,
	                        int wA, int wB, int size)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx > size)
	{
		return;
	}
	//float alpha = alphaTable[idx / 3];
	int idxIndex = idx / 3;

	int alphaIdx = idxIndex % 10;

	//printf("alpha[%d] is %f\n", idx, alpha);
	//C[idx] = A[idx] * alphaTable[alphaIdx] + B[idx] * (1 - alpha);
	float alpha = alphaIdx*1.0/wA;
	C[idx] = A[idx] * alpha + B[idx]*(1-alpha);
	//printf("��ǰ�߳�����idxΪ��%d, ��õ�alphaIdxΪ��%d, A[%d]��ֵΪ��%d, C[%d]��ֵΪ%d, alpha = %f\n", idxIndex, alphaIdx, idx, A[idx], idx, C[idx], alpha);

}

__global__ void matrixMeltVer(uchar3* A, uchar3* B, uchar3* C,
	                          int wA, int hA, int size)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx > size)
	{
		return;
	}
	//float alpha = alphaTable[idx / 3];

	int alphaIdx = idx / wA;

	//printf("alpha[%d] is %f\n", idx, alpha);
	//C[idx] = A[idx] * alphaTable[alphaIdx] + B[idx] * (1 - alpha);
	float alpha = alphaIdx * 1.0 / hA;
	// C[idx] = A[idx] * alpha + B[idx] * (1 - alpha);
	C[idx].x = A[idx].x + B[idx].x;
	C[idx].y = A[idx].y + B[idx].y;
	C[idx].z = A[idx].z + B[idx].z;
	//printf("��ǰ�߳�����idxΪ��%d, ��õ�alphaIdxΪ��%d, A[%d]��ֵΪ��%d, C[%d]��ֵΪ%d, alpha = %f\n", idxIndex, alphaIdx, idx, A[idx], idx, C[idx], alpha);

}


// \brief: ʵ��ͼ������ֱ�����ϵĽ����ںϡ�����ʵ�ֻҶ�ͼ�Ͷ�ͨ��ͼ�񡣶�ͨ����ʱ��length = wA*hA*channels
// \para: img1, img2���������ͼ��imgres�������� wA, hA�ֱ���ͼ��Ŀ�͸ߣ� length�����ݳ���
__global__ void MeltVer(u_char* img1, u_char* img2, u_char* imgres, int wA, int hA, int length) {
	int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ threadIdx.z * (blockDim.x * blockDim.y) \
		+ threadIdx.y * blockDim.x \
		+ threadIdx.x;
	float alpha = 1.0*tid /3/ wA / hA;
	if (tid < length) {
		imgres[tid] = img1[tid]*(1-alpha) + img2[tid]*(alpha);
	}
}

// \brief: ʵ��ͼ���ں������ϵĽ����ںϡ�����ʵ�ֻҶ�ͼ�Ͷ�ͨ��ͼ�񡣶�ͨ����ʱ��length = wA*hA*channels
// \para: img1, img2���������ͼ��imgres�������� wA, hA�ֱ���ͼ��Ŀ�͸ߣ� length�����ݳ���
__global__ void MeltHor(u_char* img1, u_char* img2, u_char* imgres, int wA, int hA, int length) {
	int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ threadIdx.z * (blockDim.x * blockDim.y) \
		+ threadIdx.y * blockDim.x \
		+ threadIdx.x;
	int widthLine = 3 * wA;// 3��ʾ����ͨ����
	float alpha = 1.0f * (tid %widthLine) / widthLine;
	//printf("alpha: %f\n", alpha);
	if (tid < length) {
		imgres[tid] = img1[tid]*(1-alpha) + img2[tid]*(alpha);
	}
}

// \brief: ����������uchar3���͵��ںϺ���
__global__ void MeltHor(uchar3* img1, uchar3* img2, uchar3* imgres, int wA, int hA, int length) {
	int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ threadIdx.z * (blockDim.x * blockDim.y) \
		+ threadIdx.y * blockDim.x \
		+ threadIdx.x;
	float alpha = 1.0 * tid / hA / wA;
	if (tid < length) {
		imgres[tid].x = img1[tid].x * (1 - alpha) + img2[tid].x * (alpha);
		imgres[tid].y = img1[tid].y * (1 - alpha) + img2[tid].y * (alpha);
		imgres[tid].z = img1[tid].z * (1 - alpha) + img2[tid].z * (alpha);
	}
}

// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ�ͼ��Ϊ8UC3��ʽ
// \para: mat1, mat2������Ҫ�غϵ�ͼ��,ע����GPU�ϵ����ݣ� width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ� channels��ͨ����
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
int matMeltHor(u_char * mat1, u_char * mat2, u_char * dst, int width, int height, int channels)
{
	dim3 threadsPerBlock(32, 32);
	int gridCols = (width * channels + 32 - 1) / threadsPerBlock.x;
	int gridRows = (height + 32 - 1) / threadsPerBlock.y;
	dim3 blocksPerGrid(gridCols, gridRows);
	int size = width * height * channels;
	matrixMeltHor << <blocksPerGrid, threadsPerBlock >> > (mat1, mat2, dst, width, width, size);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}

// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ������ںϵ�ͼ����8UC3��ͼ������
// \para: mat1, mat2������Ҫ�غϵ�ͼ��ע����GPU�ϵ����ݣ� width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ� channels��ͨ����
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
// NOTE:����δ�ɹ����д��Ż�
int matMeltVer(u_char* mat1, u_char* mat2, u_char* dst, int width, int height, int channels)
{
	dim3 threadsPerBlock(32, 32);
	uchar3* dev_A;
	uchar3* dev_B;
	uchar3* dev_C;
	int mallocSize = width*height*3*sizeof(u_char);
	hipMalloc((void**)&dev_A, mallocSize);
	hipMalloc((void**)&dev_B, mallocSize);
	hipMalloc((void**)&dev_C, mallocSize);
	hipMemcpy(dev_A, mat1, mallocSize, hipMemcpyDeviceToDevice);
	hipMemcpy(dev_B, mat2, mallocSize, hipMemcpyDeviceToDevice);
	// hipMemcpy(dev_A, mat1, mallocSize, hipMemcpyDeviceToDevice);
	int gridCols = (width*channels+32-1)/threadsPerBlock.x;
	int gridRows = (height+32-1)/threadsPerBlock.y;
	dim3 blocksPerGrid(gridCols, gridRows);
	int size = width * height * channels;
	matrixMeltVer<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C,  width, height, width*height);
	HANDLE_ERROR(hipGetLastError());
	hipMemcpy(dst, dev_C, mallocSize, hipMemcpyDeviceToDevice);
	return 0;
}

// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ������ںϵ�ͼ����8UC3��ͼ������
// \para: mat1, mat2������Ҫ�غϵ�ͼ��ע����GPU�ϵ�����,���ݸ�ʽע�⣻ width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ�
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
int cudaMeltVer(u_char* mat1, u_char* mat2, u_char* dst, int width, int height, int channels)
{
	int length = width*height*channels;
	dim3 gridrgb(1 + (length / (32 * 32 + 1)), 1, 1);
    dim3 blockrgb(32, 32, 1);
    MeltVer<< <gridrgb, blockrgb >> > (mat1, mat2, dst, width, height, length);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}


// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ������ںϵ�ͼ����8UC3��ͼ������
// \para: mat1, mat2������Ҫ�غϵ�ͼ��ע����GPU�ϵ�����,���ݸ�ʽע�⣻ width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ�
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
int cudaMeltHor(u_char* mat1, u_char* mat2, u_char* dst, int width, int height, int channels)
{
	int length = width*height*channels;
	dim3 gridrgb(1 + (length / (32 * 32 + 1)), 1, 1);
    dim3 blockrgb(32, 32, 1);
    MeltHor<< <gridrgb, blockrgb >> > (mat1, mat2, dst, width, height, length);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}


int mainTest()
{
	u_char A[10][30];
	u_char B[10][30];
	u_char C[10][30] = {0};
	for(int i=0;i<10;i++)
	{
		for(int j=0;j<30;j++)
		{
			A[i][j] = 100;
			B[i][j] = 0;
		}
	}
	u_char* dev_A;
	u_char* dev_B;
	u_char* dev_C;
	int width = 10; int height = 10; int channels = 3;
	float* dev_alphaTableVer = NULL;
	float* dev_alphaTableHor = NULL;
	const int mallocSize = sizeof(u_char) * width * height * channels;
	HANDLE_ERROR(hipMalloc((void**)&dev_A, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_B, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_C, mallocSize));
	int sizeOfTableHor = sizeof(float) * height;
	// HANDLE_ERROR(hipMemcpy(dev_alphaTableHor, alphaTableHor, sizeOfTableHor, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_A, A, mallocSize, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_B, B, mallocSize, hipMemcpyHostToDevice));
	matMeltVer(dev_A, dev_B, dev_C, width, height, channels);
	//dim3 threadsPerBlock(32, 32);
	//int gridCols = (width * channels + 32 - 1) / threadsPerBlock.x;
	//int gridRows = (height + 32 - 1) / threadsPerBlock.y;
	//dim3 blocksPerGrid(gridCols, gridRows);
	//int size = width * height * channels;
	//matrixMelt << <blocksPerGrid, threadsPerBlock >> > (dev_A, dev_B, dev_C, width, width, size);
	//HANDLE_ERROR(hipGetLastError());
	hipMemcpy(C, dev_C, mallocSize, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < height; i++)
	{
		for (size_t j = 0; j < width*channels; j++)
		{
			printf("%d ", C[i][j]);

		}
		printf("\n");
	}
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	return 0;
}