#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "device_types.h"
#include <assert.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "cudaMelt.cuh"
#include <opencv2/opencv.hpp>


__global__ void matrixMeltTable(unsigned char* A, unsigned char* B, unsigned char* C, int start,
int wA, int wB, int size,  float* alphaTable)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by*gridDim.x+blockIdx.x)*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x;
	if (idx > size)
	{
		return;
	}
	float alpha = alphaTable[idx/3];
	int idxIndex = idx / 3;

	int alphaIdx = idxIndex % 10;
	
	//printf("alpha[%d] is %f\n", idx, alpha);
	//C[idx] = A[idx] * alphaTable[alphaIdx] + B[idx] * (1 - alpha);
	C[idx] = A[idx]*alphaTable[alphaIdx]+ (int)B[idx];
	printf("��ǰ�߳�����idxΪ��%d, ��õ�alphaIdxΪ��%d, A[%d]��ֵΪ��%d, C[%d]��ֵΪ%d, alphaTable[alphaIdx] = %f\n", idxIndex, alphaIdx,idx, A[idx], idx, C[idx], alphaTable[alphaIdx]);
}

__global__ void matrixMeltHor(unsigned char* A, unsigned char* B, unsigned char* C,
	                        int wA, int wB, int size)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx > size)
	{
		return;
	}
	//float alpha = alphaTable[idx / 3];
	int idxIndex = idx / 3;

	int alphaIdx = idxIndex % 10;

	//printf("alpha[%d] is %f\n", idx, alpha);
	//C[idx] = A[idx] * alphaTable[alphaIdx] + B[idx] * (1 - alpha);
	float alpha = alphaIdx*1.0/wA;
	C[idx] = A[idx] * alpha + B[idx]*(1-alpha);
	//printf("��ǰ�߳�����idxΪ��%d, ��õ�alphaIdxΪ��%d, A[%d]��ֵΪ��%d, C[%d]��ֵΪ%d, alpha = %f\n", idxIndex, alphaIdx, idx, A[idx], idx, C[idx], alpha);

}

__global__ void matrixMeltVer(uchar3* A, uchar3* B, uchar3* C,
	                          int wA, int hA, int size)
{
	// block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int idx = (by * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (idx > size)
	{
		return;
	}
	//float alpha = alphaTable[idx / 3];

	int alphaIdx = idx / wA;

	//printf("alpha[%d] is %f\n", idx, alpha);
	//C[idx] = A[idx] * alphaTable[alphaIdx] + B[idx] * (1 - alpha);
	float alpha = alphaIdx * 1.0 / hA;
	// C[idx] = A[idx] * alpha + B[idx] * (1 - alpha);
	C[idx].x = A[idx].x + B[idx].x;
	C[idx].y = A[idx].y + B[idx].y;
	C[idx].z = A[idx].z + B[idx].z;
	//printf("��ǰ�߳�����idxΪ��%d, ��õ�alphaIdxΪ��%d, A[%d]��ֵΪ��%d, C[%d]��ֵΪ%d, alpha = %f\n", idxIndex, alphaIdx, idx, A[idx], idx, C[idx], alpha);

}


// \brief: ʵ��ͼ������ֱ�����ϵĽ����ںϡ�����ʵ�ֻҶ�ͼ�Ͷ�ͨ��ͼ�񡣶�ͨ����ʱ��length = wA*hA*channels
// \para: img1, img2���������ͼ��imgres�������� wA, hA�ֱ���ͼ��Ŀ�͸ߣ� length�����ݳ���
__global__ void MeltVer(cv::cuda::PtrStepSz<uchar> img1, cv::cuda::PtrStepSz<uchar>  img2, cv::cuda::PtrStepSz<uchar>  imgres, int wA, int hA, int length) {
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	if (tx >= length ||ty >= hA)
	{
		return;
	}
	float alpha = 1.0*ty/hA;
	int did = ty*imgres.step + tx;
	int sid1= ty*img1.step + tx;
	int sid2 = ty*img2.step + tx;
	imgres[did] = img1[sid1]*(1-alpha) + img2[sid2]*(alpha);
		// printf("%d   ", tid);
	
}

// \brief: ʵ��ͼ���ں������ϵĽ����ںϡ�����ʵ�ֻҶ�ͼ�Ͷ�ͨ��ͼ�񡣶�ͨ����ʱ��length = wA*hA*channels
// \para: img1, img2���������ͼ��imgres�������� wA, hA�ֱ���ͼ��Ŀ�͸ߣ� length�����ݳ���
__global__ void MeltHor(cv::cuda::PtrStepSz<uchar> img1, cv::cuda::PtrStepSz<uchar>  img2, cv::cuda::PtrStepSz<uchar>  imgres, int wA, int hA, int length) {
	int tx = blockIdx.x * blockDim.x + threadIdx.x;
	int ty = blockIdx.y * blockDim.y + threadIdx.y;
	if (tx >= length || ty >= hA)
	{
		return;
	}
	float alpha = 1.0 * tx /3/ wA;
	int did = ty * imgres.step + tx;
	int sid1 = ty * img1.step + tx;
	int sid2 = ty * img2.step + tx;
	imgres[did] = img1[sid1] * (1 - alpha) + img2[sid2] * (alpha);
	// printf("%d   ", tid);
}

// \brief: ����������uchar3���͵��ںϺ���
// __global__ void MeltHorU3(uchar3* img1, uchar3* img2, uchar3* imgres, int wA, int hA, int length) {
__global__ void MeltHorU3(cv::cuda::PtrStepSz<uchar3> img1, cv::cuda::PtrStepSz<uchar3>  img2, cv::cuda::PtrStep<uchar3> imgres, int wA, int hA, int length, int step) {
	int tid = blockIdx.z * (gridDim.x * gridDim.y) * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.y * gridDim.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ blockIdx.x * (blockDim.x * blockDim.y * blockDim.z) \
		+ threadIdx.z * (blockDim.x * blockDim.y) \
		+ threadIdx.y * blockDim.x \
		+ threadIdx.x;
	float alpha = 1.0 * tid / hA / wA;
	int steps = img1.step;
	int tx = threadIdx.x + blockIdx.x*blockDim.x;
	int ty = threadIdx.y + blockIdx.y*blockDim.y;
	if (tx < wA && ty<hA) {
		int row = tid / wA;
		// did ��ʾ�������ݵ���������
		// int did = (row*steps)/sizeof(uchar3) + tid % wA;
		// printf("Step is :%d\n", steps);
		int did = ty*steps/3 + tx;
		int rid = ty * imgres.step/3 + tx;
		imgres.data[rid].z = img1.data[did].z * (1 - alpha) + img2.data[did].z * (alpha);
		imgres.data[rid].x = img1.data[did].x * (1 - alpha) + img2.data[did].x * (alpha);
		imgres.data[rid].y = img1.data[did].y * (1 - alpha) + img2.data[did].y * (alpha);
	}
}

// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ�ͼ��Ϊ8UC3��ʽ
// \para: mat1, mat2������Ҫ�غϵ�ͼ��,ע����GPU�ϵ����ݣ� width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ� channels��ͨ����
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
int matMeltHor(u_char * mat1, u_char * mat2, u_char * dst, int width, int height, int channels)
{
	dim3 threadsPerBlock(32, 32);
	int gridCols = (width * channels + 32 - 1) / threadsPerBlock.x;
	int gridRows = (height + 32 - 1) / threadsPerBlock.y;
	dim3 blocksPerGrid(gridCols, gridRows);
	int size = width * height * channels;
	matrixMeltHor << <blocksPerGrid, threadsPerBlock >> > (mat1, mat2, dst, width, width, size);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}

// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ������ںϵ�ͼ����8UC3��ͼ������
// \para: mat1, mat2������Ҫ�غϵ�ͼ��ע����GPU�ϵ����ݣ� width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ� channels��ͨ����
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
// NOTE:����δ�ɹ����д��Ż�
int matMeltVer(u_char* mat1, u_char* mat2, u_char* dst, int width, int height, int channels)
{
	dim3 threadsPerBlock(32, 32);
	uchar3* dev_A;
	uchar3* dev_B;
	uchar3* dev_C;
	int mallocSize = width*height*3*sizeof(u_char);
	hipMalloc((void**)&dev_A, mallocSize);
	hipMalloc((void**)&dev_B, mallocSize);
	hipMalloc((void**)&dev_C, mallocSize);
	hipMemcpy(dev_A, mat1, mallocSize, hipMemcpyDeviceToDevice);
	hipMemcpy(dev_B, mat2, mallocSize, hipMemcpyDeviceToDevice);
	// hipMemcpy(dev_A, mat1, mallocSize, hipMemcpyDeviceToDevice);
	int gridCols = (width*channels+32-1)/threadsPerBlock.x;
	int gridRows = (height+32-1)/threadsPerBlock.y;
	dim3 blocksPerGrid(gridCols, gridRows);
	int size = width * height * channels;
	matrixMeltVer<<<blocksPerGrid, threadsPerBlock>>>(dev_A, dev_B, dev_C,  width, height, width*height);
	HANDLE_ERROR(hipGetLastError());
	hipMemcpy(dst, dev_C, mallocSize, hipMemcpyDeviceToDevice);
	return 0;
}

// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ������ںϵ�ͼ����8UC3��ͼ������
// \para: mat1, mat2������Ҫ�غϵ�ͼ��ע����GPU�ϵ�����,���ݸ�ʽע�⣻ width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ�
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
int cudaMeltVer(cv::cuda::GpuMat mat1, cv::cuda::GpuMat mat2, cv::cuda::GpuMat dst, int width, int height, int channels)
{
	int length = width * channels;
	dim3 blockrgb(32, 32);
	dim3 gridrgb((length + blockrgb.x - 1) / blockrgb.x, (height + blockrgb.y - 1) / blockrgb.y);
	MeltVer << <gridrgb, blockrgb >> > (mat1, mat2, dst, width, height, length);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}


// \brief: ʵ������ͼ�������غϲ��ֵ��ںϣ�����������ͼ����ȫ�ںϡ������ںϵ�ͼ����8UC3��ͼ������
// \para: mat1, mat2������Ҫ�غϵ�ͼ��ע����GPU�ϵ�����,���ݸ�ʽע�⣻ width��height�ֱ���ͼ��Ŀ�Ⱥ͸߶ȣ�
// �����blocksizeĬ��������32��������Ը���ʵ��blockSize�������ã�
int cudaMeltHor(cv::cuda::GpuMat mat1, cv::cuda::GpuMat mat2, cv::cuda::GpuMat dst, int width, int height, int channels)
{
	int length = width*channels;
    dim3 blockrgb(32, 32);
	dim3 gridrgb((length + blockrgb.x - 1) / blockrgb.x, (height + blockrgb.y - 1) / blockrgb.y);
    MeltHor<< <gridrgb, blockrgb >> > (mat1, mat2, dst, width, height, length);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}

int cudaMeltHorUC3(cv::cuda::GpuMat mat1, cv::cuda::GpuMat mat2, cv::cuda::GpuMat dst, int width, int height, int channels)
{
	int length = width*height;
    dim3 blockrgb(32, 32);
	dim3 gridrgb((width+blockrgb.x-1)/blockrgb.x , (height + blockrgb.y-1)/blockrgb.y);
	MeltHorU3<<<gridrgb, blockrgb>>>(mat1, mat2, dst, width, height, length, mat1.step);
	HANDLE_ERROR(hipGetLastError());
	return 0;
}


int maincu()
{
	u_char A[30][10];
	u_char B[30][10];
	u_char C[30][10] = {0};
	for(int i=0;i<10;i++)
	{
		for(int j=0;j<30;j++)
		{
			A[i][j] = 100;
			B[i][j] = 0;
		}
	}
	u_char* dev_A;
	u_char* dev_B;
	u_char* dev_C;
	int width = 10; int height = 10; int channels = 3;
	float* dev_alphaTableVer = NULL;
	float* dev_alphaTableHor = NULL;
	int mallocSize = sizeof(u_char) * width * height*3;
	HANDLE_ERROR(hipMalloc((void**)&dev_A, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_B, mallocSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_C, mallocSize));
	int sizeOfTableHor = sizeof(float) * height;
	// HANDLE_ERROR(hipMemcpy(dev_alphaTableHor, alphaTableHor, sizeOfTableHor, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_A, A, mallocSize, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_B, B, mallocSize, hipMemcpyHostToDevice));
	// cudaMeltHor(dev_A, dev_B, dev_C, width, height, channels);
	//dim3 threadsPerBlock(32, 32);
	//int gridCols = (width * channels + 32 - 1) / threadsPerBlock.x;
	//int gridRows = (height + 32 - 1) / threadsPerBlock.y;
	//dim3 blocksPerGrid(gridCols, gridRows);
	//int size = width * height * channels;
	//matrixMelt << <blocksPerGrid, threadsPerBlock >> > (dev_A, dev_B, dev_C, width, width, size);
	//HANDLE_ERROR(hipGetLastError());
	// hipMemcpy(C, dev_C, mallocSize, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < height; i++)
	{
		for (size_t j = 0; j < width*channels; j++)
		{
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	return 0;
}